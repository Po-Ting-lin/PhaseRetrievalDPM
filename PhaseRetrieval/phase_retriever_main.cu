#include "hip/hip_runtime.h"
#include "phase_retriever.cuh"

#define DEBUG false
#define TIMER false

void PhaseRetriever(uchar* sp, uchar* bg, float* dst, int width, int height, int spx, int spy, int bgx, int bgy) {
	PhaseRetrieverInfo info;
	info.Image = nullptr;
	info.WrappedImage = nullptr;
	info.UnwrappedImage = nullptr;
	info.Dst = dst;
	info.Width = width;
	info.Height = height;
	info.CroppedWidth = width / 4;
	info.CroppedHeight = height / 4;
	info.CroppedSPOffsetX = spx;
	info.CroppedSPOffsetY = spy;
	info.CroppedBGOffsetX = bgx;
	info.CroppedBGOffsetY = bgy;
	info.NumberOfRealElements = width * height;
	info.NumberOfCropElements = (width / 4) * (height / 4);
	info.DataElementsPerStream = info.NumberOfRealElements / D_NUM_STREAMS;
	info.DataBytesPerStream = info.NumberOfRealElements * sizeof(uchar) / D_NUM_STREAMS;
	info.Blocks = new dim3(TILE_DIM, TILE_DIM);
	info.Blocks1D = new dim3(TILE_DIM * 2); // occupancy will be 100% for dim.x = 64
	info.Grids = new dim3(iDivUp(width, TILE_DIM), iDivUp(height, TILE_DIM));
	info.CroppedGrids = new dim3(iDivUp(width / 4, TILE_DIM), iDivUp(height / 4, TILE_DIM));
	info.Grids1D = new dim3(iDivUp(info.DataElementsPerStream, TILE_DIM * 2));
	
	float* sp_unwarpped = nullptr;
	float* bg_unwarpped = nullptr;
	imageRetriever(sp, sp_unwarpped, info, true);
	imageRetriever(bg, bg_unwarpped, info, false);

	for (int i = 0; i < info.NumberOfCropElements; i++) {
		sp_unwarpped[i] -= bg_unwarpped[i];
	}
	
	free(bg_unwarpped);
	delete info.Blocks;
	delete info.Blocks1D;
	delete info.Grids;
	delete info.Grids1D;
	delete info.CroppedGrids;
}

void imageRetriever(uchar* src, float*& dst, PhaseRetrieverInfo& info, bool isSp) {
	info.Image = src;
#if TIMER
	auto t0 = std::chrono::system_clock::now();
#endif
	getWrappedImage(info, isSp);
#if TIMER
	auto t1 = std::chrono::system_clock::now();
#endif
	getUnwrappedImage(info, isSp);
#if TIMER
	auto t2 = std::chrono::system_clock::now();
	printTime(t0, t1, "getWrappedImage");
	printTime(t1, t2, "getUnwrappedImage");
#endif
	dst = info.UnwrappedImage;
}

void getWrappedImage(PhaseRetrieverInfo& info, bool isSp) {
	float* d_magnitude;
	float* d_firstCrop;
	uchar* image_ptr = info.Image;
	uchar* d_image_ptr;
	fComplex* d_data;
	fComplex* d_Spectrum;
	fComplex* d_SecondCrop;
	fComplex* d_rawWrapped;

	// make a FFT plan
	hipfftHandle fftPlan;
	hipfftHandle ifftPlan;
	gpuErrorCheck(hipfftPlan2d(&fftPlan, info.Height, info.Width, HIPFFT_C2C));
	gpuErrorCheck(hipfftPlan2d(&ifftPlan, info.CroppedWidth, info.CroppedHeight, HIPFFT_C2C));

	// aysn H to D
	hipStream_t stream[D_NUM_STREAMS];
	for (int i = 0; i < D_NUM_STREAMS; i++) {
		hipStreamCreate(&stream[i]);
	}
	gpuErrorCheck(hipMalloc((uchar**)&d_image_ptr, info.NumberOfRealElements * sizeof(uchar)));
	gpuErrorCheck(hipMalloc((fComplex**)&d_data, info.NumberOfRealElements * sizeof(fComplex)));
	int offset = 0;
	for (int i = 0; i < D_NUM_STREAMS; i++) {
		offset = i * info.DataElementsPerStream;
		gpuErrorCheck(hipMemcpyAsync(&d_image_ptr[offset], &image_ptr[offset], info.DataBytesPerStream, hipMemcpyHostToDevice, stream[i]));
		realToComplex << <*info.Grids1D, *info.Blocks1D, 0, stream[i] >> > (&d_image_ptr[offset], &d_data[offset], info.NumberOfRealElements);
	}
	gpuErrorCheck(hipDeviceSynchronize());
	for (int i = 0; i < D_NUM_STREAMS; i++) {
		gpuErrorCheck(hipStreamDestroy(stream[i]));
	}

	// fft
	gpuErrorCheck(hipMalloc((fComplex**)&d_Spectrum, info.NumberOfRealElements * sizeof(fComplex)));
	gpuErrorCheck(hipfftExecC2C(fftPlan, (hipfftComplex*)d_data, (hipfftComplex*)d_Spectrum, HIPFFT_FORWARD));
	gpuErrorCheck(hipDeviceSynchronize());

	// magnitude
	gpuErrorCheck(hipMalloc((float**)&d_magnitude, info.NumberOfRealElements * sizeof(float)));
	complexToMagnitude << <*info.Grids, *info.Blocks >> > (d_Spectrum, d_magnitude, info.Width, info.Height);
	gpuErrorCheck(hipDeviceSynchronize());

#if DEBUG
	float* h_magnitude = (float*)malloc(info.NumberOfRealElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_magnitude, d_magnitude, info.NumberOfRealElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_magnitude, info.Width, info.Height, true);
#endif

	// find max index
	gpuErrorCheck(hipMalloc((float**)&d_firstCrop, info.NumberOfCropElements * sizeof(float)));
	copyInterferenceComponentRoughly << <*info.CroppedGrids, *info.Blocks >> > (d_magnitude, d_firstCrop, info.Width, info.Height, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

#if DEBUG
	float* h_firstCrop = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_firstCrop, d_firstCrop, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_firstCrop, info.CroppedWidth, info.CroppedHeight, true);
#endif

	thrust::device_ptr<float> d_ptr(d_firstCrop);
	thrust::device_vector<float> d_vec(d_ptr, d_ptr + info.NumberOfCropElements);
	thrust::device_vector<float>::iterator iter = thrust::max_element(d_vec.begin(), d_vec.end());
	unsigned int index = iter - d_vec.begin();
	float max_val = *iter;
	int max_cropped_x = index % info.CroppedWidth;
	int max_cropped_y = index / info.CroppedWidth;
	int offset_x = isSp ? info.CroppedSPOffsetX : info.CroppedBGOffsetX;
	int offset_y = isSp ? info.CroppedSPOffsetY : info.CroppedBGOffsetY;
	int max_x = max_cropped_x < info.CroppedWidth / 2 ? max_cropped_x + 7 * info.Width / 8 : max_cropped_x - info.CroppedWidth / 2 + offset_x;
	int max_y = max_cropped_y + info.Height / 2 + offset_y;

#if DEBUG
	float* d_SecondCropDebug;
	gpuErrorCheck(hipMalloc((float**)&d_SecondCropDebug, info.NumberOfCropElements * sizeof(float)));
	copyInterferenceComponentDebug << <*info.CroppedGrids, *info.Blocks >> > (d_magnitude, d_SecondCropDebug, max_x, max_y, info.Width, info.Height, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	float* h_SecondCropDebug = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_SecondCropDebug, d_SecondCropDebug, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_SecondCropDebug, info.CroppedWidth, info.CroppedHeight, false);
#endif
	gpuErrorCheck(hipMalloc((fComplex**)&d_SecondCrop, info.NumberOfCropElements * sizeof(fComplex)));
	copyInterferenceComponent << <*info.CroppedGrids, *info.Blocks >> > (d_Spectrum, d_SecondCrop, max_x, max_y, info.Width, info.Height, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	// ifft
	gpuErrorCheck(hipMalloc((fComplex**)&d_rawWrapped, info.NumberOfCropElements * sizeof(fComplex)));
	gpuErrorCheck(hipfftExecC2C(ifftPlan, (hipfftComplex*)d_SecondCrop, (hipfftComplex*)d_rawWrapped, HIPFFT_BACKWARD));
	gpuErrorCheck(hipDeviceSynchronize());

	// arctan
	gpuErrorCheck(hipMalloc((float**)&info.WrappedImage, info.NumberOfCropElements * sizeof(float)));
	applyArcTan << <*info.CroppedGrids, *info.Blocks >> > (d_rawWrapped, info.WrappedImage, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	// free
	gpuErrorCheck(hipfftDestroy(fftPlan));
	gpuErrorCheck(hipfftDestroy(ifftPlan));
	gpuErrorCheck(hipFree(d_data));
	gpuErrorCheck(hipFree(d_magnitude));
	gpuErrorCheck(hipFree(d_firstCrop));
	gpuErrorCheck(hipFree(d_SecondCrop));
	gpuErrorCheck(hipFree(d_rawWrapped));
}

void getUnwrappedImage(PhaseRetrieverInfo& info, bool isSp) {
#if DEBUG
	float* h_wrapped_image = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_wrapped_image, info.WrappedImage, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_wrapped_image, info.CroppedWidth, info.CroppedHeight, false);
	free(h_wrapped_image);
#endif
	float* h_sumC;
	float* d_dx;
	float* d_dy;
	float* d_sumC;
	float* d_divider;
	float tx = PI / info.CroppedWidth;
	float ty = PI / info.CroppedHeight;
	gpuErrorCheck(hipMalloc((float**)&d_dx, info.NumberOfCropElements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_dy, info.NumberOfCropElements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_sumC, info.NumberOfCropElements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_divider, info.NumberOfCropElements * sizeof(float)));

	// diff
	applyDifference<<<*info.CroppedGrids, *info.Blocks >>>(info.WrappedImage, d_dx, d_dy, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());
	applySum << <*info.CroppedGrids, *info.Blocks >> > (d_dx, d_dy, d_sumC, d_divider, tx, ty, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());
	gpuErrorCheck(hipFree(d_dx));
	gpuErrorCheck(hipFree(d_dy));

	if (isSp) 
		h_sumC = info.Dst;
	else 
		h_sumC = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_sumC, d_sumC, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	float* h_divider = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_divider, d_divider, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));

	cv::Mat sumC_mat(info.CroppedHeight, info.CroppedWidth, CV_32FC1, h_sumC);
	cv::dct(sumC_mat, sumC_mat);
	for (int i = 0; i < info.NumberOfCropElements; i++) {
		if (h_divider[i] != 0.0f) h_sumC[i] = h_sumC[i] / h_divider[i];
	}
	cv::idct(sumC_mat, sumC_mat);
	info.UnwrappedImage = h_sumC;

	// free
	free(h_divider);
	gpuErrorCheck(hipFree(d_sumC));
	gpuErrorCheck(hipFree(d_divider));
	gpuErrorCheck(hipFree(info.WrappedImage));
}
