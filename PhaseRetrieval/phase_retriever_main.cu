#include "hip/hip_runtime.h"
#include "phase_retriever.cuh"

#define DEBUG false
#define TIMER true

void PhaseRetriever(uchar* sp, uchar* bg, float* dst, float* spectrumDst, int width, int height, int spx, int spy, int bgx, int bgy) {
	PhaseRetrieverInfo info;
	info.Image = nullptr;
	info.WrappedImage = nullptr;
	info.UnwrappedImage = nullptr;
	info.Dst = dst;
	info.SpectrumDst = spectrumDst;
	info.Width = width;
	info.Height = height;
	info.CroppedWidth = width / 4;
	info.CroppedHeight = height / 4;
	info.CroppedSPOffsetX = spx;
	info.CroppedSPOffsetY = spy;
	info.CroppedBGOffsetX = bgx;
	info.CroppedBGOffsetY = bgy;
	info.NumberOfRealElements = width * height;
	info.NumberOfCropElements = (width / 4) * (height / 4);
	info.DataElementsPerStream = info.NumberOfRealElements / D_NUM_STREAMS;
	info.DataBytesPerStream = info.NumberOfRealElements * sizeof(uchar) / D_NUM_STREAMS;
	info.Blocks = new dim3(TILE_DIM, TILE_DIM);
	info.Blocks1D = new dim3(TILE_DIM * 2); // occupancy will be 100% for dim.x = 64
	info.Grids = new dim3(iDivUp(width, TILE_DIM), iDivUp(height, TILE_DIM));
	info.CroppedGrids = new dim3(iDivUp(width / 4, TILE_DIM), iDivUp(height / 4, TILE_DIM));
	info.Grids1D = new dim3(iDivUp(info.DataElementsPerStream, TILE_DIM * 2));
	gpuErrorCheck(hipfftPlan2d(&info.fftHandle, info.Height, info.Width, HIPFFT_C2C));
	gpuErrorCheck(hipfftPlan2d(&info.ifftHandle, info.CroppedWidth, info.CroppedHeight, HIPFFT_C2C));

	float* sp_unwarpped = nullptr;
	float* bg_unwarpped = nullptr;
	
#if TIMER
	auto t0 = std::chrono::system_clock::now();
#endif
	imageRetriever(sp, sp_unwarpped, info, true);
	imageRetriever(bg, bg_unwarpped, info, false);
	for (int i = 0; i < info.NumberOfCropElements; i++) {
		sp_unwarpped[i] -= bg_unwarpped[i];
	}
#if TIMER
	auto t1 = std::chrono::system_clock::now();
	printTime(t0, t1, "total elapsed time");
#endif
	
	free(bg_unwarpped);
	delete info.Blocks;
	delete info.Blocks1D;
	delete info.Grids;
	delete info.Grids1D;
	delete info.CroppedGrids;
}

void imageRetriever(uchar* src, float*& dst, PhaseRetrieverInfo& info, bool isSp) {
	info.Image = src;
#if TIMER
	auto t0 = std::chrono::system_clock::now();
#endif
	getWrappedImage(info, isSp);
#if TIMER
	auto t1 = std::chrono::system_clock::now();
#endif
	getUnwrappedImage(info, isSp);
#if TIMER
	auto t2 = std::chrono::system_clock::now();
	printTime(t0, t1, "getWrappedImage");
	printTime(t1, t2, "getUnwrappedImage");
#endif
	dst = info.UnwrappedImage;
}

void getWrappedImage(PhaseRetrieverInfo& info, bool isSp) {
	float* d_magnitude;
	float* d_firstCrop;
	uchar* image_ptr = info.Image;
	uchar* d_image_ptr;
	fComplex* d_data;
	fComplex* d_Spectrum;
	fComplex* d_SecondCrop;
	fComplex* d_rawWrapped;
	float* d_blk_vals;
	int* d_blk_idx;
	int* d_blk_num;
	int* d_max_index;
	int max_index = 0;
	const int block_size_1d = TILE_DIM * TILE_DIM;
	const int grid_size_1d = MIN(MAX_KERNEL_BLOCKS, ((info.NumberOfCropElements + block_size_1d - 1) / block_size_1d));
	const int max_block = (info.NumberOfCropElements / block_size_1d) + 1;

	// allocate the resources
	gpuErrorCheck(hipMalloc((uchar**)&d_image_ptr, info.NumberOfRealElements * sizeof(uchar)));
	gpuErrorCheck(hipMalloc((fComplex**)&d_data, info.NumberOfRealElements * sizeof(fComplex)));
	gpuErrorCheck(hipMalloc((fComplex**)&d_Spectrum, info.NumberOfRealElements * sizeof(fComplex)));
	gpuErrorCheck(hipMalloc((float**)&d_magnitude, info.NumberOfRealElements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_firstCrop, info.NumberOfCropElements * sizeof(float)));
	gpuErrorCheck(hipMalloc(&d_blk_vals, max_block * sizeof(float)));
	gpuErrorCheck(hipMalloc(&d_blk_idx, max_block * sizeof(int)));
	gpuErrorCheck(hipMalloc(&d_blk_num, 1 * sizeof(int)));
	gpuErrorCheck(hipMalloc(&d_max_index, sizeof(int)));
	gpuErrorCheck(hipMalloc((fComplex**)&d_SecondCrop, info.NumberOfCropElements * sizeof(fComplex)));
	gpuErrorCheck(hipMalloc((fComplex**)&d_rawWrapped, info.NumberOfCropElements * sizeof(fComplex)));
	gpuErrorCheck(hipMalloc((float**)&info.WrappedImage, info.NumberOfCropElements * sizeof(float)));

	// aysn H to D
	int offset = 0;
	hipStream_t stream[D_NUM_STREAMS];
	for (int i = 0; i < D_NUM_STREAMS; i++) {
		hipStreamCreate(&stream[i]);
	}
	
	for (int i = 0; i < D_NUM_STREAMS; i++) {
		offset = i * info.DataElementsPerStream;
		gpuErrorCheck(hipMemcpyAsync(&d_image_ptr[offset], &image_ptr[offset], info.DataBytesPerStream, hipMemcpyHostToDevice, stream[i]));
		realToComplex << <*info.Grids1D, *info.Blocks1D, 0, stream[i] >> > (&d_image_ptr[offset], &d_data[offset], info.NumberOfRealElements);
	}
	gpuErrorCheck(hipDeviceSynchronize());
	for (int i = 0; i < D_NUM_STREAMS; i++) {
		gpuErrorCheck(hipStreamDestroy(stream[i]));
	}
	gpuErrorCheck(hipfftExecC2C(info.fftHandle, (hipfftComplex*)d_data, (hipfftComplex*)d_Spectrum, HIPFFT_FORWARD));
	gpuErrorCheck(hipDeviceSynchronize());
	complexToMagnitude << <*info.Grids, *info.Blocks >> > (d_Spectrum, d_magnitude, info.Width, info.Height);
	gpuErrorCheck(hipDeviceSynchronize());

#if DEBUG
	float* h_magnitude = (float*)malloc(info.NumberOfRealElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_magnitude, d_magnitude, info.NumberOfRealElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_magnitude, info.Width, info.Height, true);
#endif

	if (isSp) {
		gpuErrorCheck(hipMemcpy(info.SpectrumDst, d_magnitude, info.NumberOfRealElements * sizeof(float), hipMemcpyDeviceToHost));
	}

	// find max index
	copyInterferenceComponentRoughly << <*info.CroppedGrids, *info.Blocks >> > (d_magnitude, d_firstCrop, info.Width, info.Height, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

#if DEBUG
	float* h_firstCrop = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_firstCrop, d_firstCrop, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_firstCrop, info.CroppedWidth, info.CroppedHeight, true);
#endif

	hipMemset(d_blk_vals, 0.0f, max_block * sizeof(float));
	hipMemset(d_blk_idx, 0, max_block * sizeof(int));
	hipMemset(d_blk_num, 0, 1 * sizeof(int));
	max_idx_kernel << <grid_size_1d, block_size_1d >> > (d_firstCrop, info.NumberOfCropElements, d_max_index, d_blk_vals, d_blk_idx, d_blk_num);
	hipMemcpy(&max_index, d_max_index, sizeof(int), hipMemcpyDeviceToHost);

	int max_cropped_x = max_index % info.CroppedWidth;
	int max_cropped_y = max_index / info.CroppedWidth;
	int offset_x = isSp ? info.CroppedSPOffsetX : info.CroppedBGOffsetX;
	int offset_y = isSp ? info.CroppedSPOffsetY : info.CroppedBGOffsetY;
	int max_x = max_cropped_x < info.CroppedWidth / 2 ? max_cropped_x + 7 * info.Width / 8 : max_cropped_x - info.CroppedWidth / 2 + offset_x;
	int max_y = max_cropped_y + info.Height / 2 + offset_y;

#if DEBUG
	float* d_SecondCropDebug;
	gpuErrorCheck(hipMalloc((float**)&d_SecondCropDebug, info.NumberOfCropElements * sizeof(float)));
	copyInterferenceComponentDebug << <*info.CroppedGrids, *info.Blocks >> > (d_magnitude, d_SecondCropDebug, max_x, max_y, info.Width, info.Height, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	float* h_SecondCropDebug = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_SecondCropDebug, d_SecondCropDebug, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_SecondCropDebug, info.CroppedWidth, info.CroppedHeight, false);
#endif
	copyInterferenceComponent << <*info.CroppedGrids, *info.Blocks >> > (d_Spectrum, d_SecondCrop, max_x, max_y, info.Width, info.Height, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	// ifft
	gpuErrorCheck(hipfftExecC2C(info.ifftHandle, (hipfftComplex*)d_SecondCrop, (hipfftComplex*)d_rawWrapped, HIPFFT_BACKWARD));
	gpuErrorCheck(hipDeviceSynchronize());

	// arctan
	applyArcTan << <*info.CroppedGrids, *info.Blocks >> > (d_rawWrapped, info.WrappedImage, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	// free
	gpuErrorCheck(hipFree(d_blk_vals));
	gpuErrorCheck(hipFree(d_blk_idx));
	gpuErrorCheck(hipFree(d_blk_num));
	gpuErrorCheck(hipFree(d_max_index));
	gpuErrorCheck(hipFree(d_data));
	gpuErrorCheck(hipFree(d_magnitude));
	gpuErrorCheck(hipFree(d_firstCrop));
	gpuErrorCheck(hipFree(d_SecondCrop));
	gpuErrorCheck(hipFree(d_rawWrapped));
}

void getUnwrappedImage(PhaseRetrieverInfo& info, bool isSp) {
#if DEBUG
	float* h_wrapped_image = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_wrapped_image, info.WrappedImage, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_wrapped_image, info.CroppedWidth, info.CroppedHeight, false);
	free(h_wrapped_image);
#endif
	float* h_sumC;
	float* d_dx;
	float* d_dy;
	float* d_sumC;
	float* d_divider;
	float tx = PI / info.CroppedWidth;
	float ty = PI / info.CroppedHeight;
	gpuErrorCheck(hipMalloc((float**)&d_dx, info.NumberOfCropElements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_dy, info.NumberOfCropElements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_sumC, info.NumberOfCropElements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_divider, info.NumberOfCropElements * sizeof(float)));

	// diff
	applyDifference<<<*info.CroppedGrids, *info.Blocks >>>(info.WrappedImage, d_dx, d_dy, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());
	applySum << <*info.CroppedGrids, *info.Blocks >> > (d_dx, d_dy, d_sumC, d_divider, tx, ty, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());
	gpuErrorCheck(hipFree(d_dx));
	gpuErrorCheck(hipFree(d_dy));

	if (isSp) 
		h_sumC = info.Dst;
	else 
		h_sumC = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_sumC, d_sumC, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	float* h_divider = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_divider, d_divider, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));

	cv::Mat sumC_mat(info.CroppedHeight, info.CroppedWidth, CV_32FC1, h_sumC);
	cv::dct(sumC_mat, sumC_mat);
	for (int i = 0; i < info.NumberOfCropElements; i++) {
		if (h_divider[i] != 0.0f) h_sumC[i] = h_sumC[i] / h_divider[i];
	}
	cv::idct(sumC_mat, sumC_mat);
	info.UnwrappedImage = h_sumC;

	// free
	free(h_divider);
	gpuErrorCheck(hipFree(d_sumC));
	gpuErrorCheck(hipFree(d_divider));
	gpuErrorCheck(hipFree(info.WrappedImage));
}
