#include "hip/hip_runtime.h"
#include "phase_retriever.cuh"
#include <hipfft/hipfft.h>

#define DEBUG false

void processPhaseRetriever(cv::Mat& src) {
	PhaseRetrieverInfo info;
	info.Image = &src;
	info.Width = src.cols;
	info.HalfWidth = (src.cols / 2 + 1);
	info.Height = src.rows;
	info.NumberOfRealElements = src.cols * src.rows;
	info.NumberOfComplexElements = (src.cols / 2 + 1) * src.rows;
	getWrappedImage(info);
	getUnwrappedImage();
}

void getWrappedImage(PhaseRetrieverInfo& info) {
	// fft
	uchar* image_ptr = info.Image->data;
	float* d_data;
	fComplex* d_Spectrum;

	float* h_data = new float[info.Width * info.Height];
	for (int i = 0; i < info.Width * info.Height; i++) {
		h_data[i] = image_ptr[i];
	}

	hipfftHandle fftPlan;
	long src_byte_size = info.NumberOfRealElements * sizeof(float);
	gpuErrorCheck(hipMalloc((float**)&d_data, src_byte_size));
	gpuErrorCheck(hipMalloc((void**)&d_Spectrum, info.NumberOfComplexElements * sizeof(fComplex)));
	gpuErrorCheck(hipMemcpy(d_data, h_data, src_byte_size, hipMemcpyHostToDevice));
	gpuErrorCheck(hipfftPlan2d(&fftPlan, info.Height, info.Width, HIPFFT_R2C));
	gpuErrorCheck(hipfftExecR2C(fftPlan, (hipfftReal*)d_data, (hipfftComplex*)d_Spectrum));
	gpuErrorCheck(hipDeviceSynchronize());
	gpuErrorCheck(hipFree(d_data));


	// magnitude

	float* d_magnitude;
	gpuErrorCheck(hipMalloc((float**)&d_magnitude, info.NumberOfComplexElements * sizeof(float)));
	dim3 blocks(TILE_DIM, TILE_DIM);
	dim3 grids(iDivUp(info.HalfWidth, TILE_DIM), iDivUp(info.Height, TILE_DIM));
	complexToMagnitude << <grids, blocks >> > (d_Spectrum, d_magnitude, info.HalfWidth, info.Height);
	gpuErrorCheck(hipDeviceSynchronize());

#if DEBUG
	float* h_magnitude = (float*)malloc(info.NumberOfComplexElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_magnitude, d_magnitude, info.NumberOfComplexElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_magnitude, info.HalfWidth, info.Height, true);
#endif

	// find max index
	float* d_firstCrop;
	int frist_cropped_width = info.Width / 8;
	int first_cropped_height = info.Height / 2;
	gpuErrorCheck(hipMalloc((float**)&d_firstCrop, frist_cropped_width * first_cropped_height * sizeof(float)));
	copyInterferenceComponentRoughly << <grids, blocks >> > (d_magnitude, d_firstCrop, info.Height / 4, frist_cropped_width, first_cropped_height, info.HalfWidth, info.Height, frist_cropped_width);
	gpuErrorCheck(hipDeviceSynchronize());

#if DEBUG
	float* h_firstCrop = (float*)malloc(first_cropped_height * frist_cropped_width * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_firstCrop, d_firstCrop, first_cropped_height * frist_cropped_width * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_firstCrop, frist_cropped_width, first_cropped_height, true);
#endif

	thrust::device_ptr<float> d_ptr(d_firstCrop);
	thrust::device_vector<float> d_vec(d_ptr, d_ptr + frist_cropped_width * first_cropped_height);
	thrust::device_vector<float>::iterator iter = thrust::max_element(d_vec.begin(), d_vec.end());
	unsigned int index = iter - d_vec.begin();
	float max_val = *iter;
	int max_loc_x = index % frist_cropped_width;
	int max_loc_y = index / frist_cropped_width + info.Height / 4;
	std::cout << "Position is x: " << max_loc_x << "y: "<< max_loc_y << std::endl;
	gpuErrorCheck(hipFree(d_firstCrop));


	// crop
	int second_cropped_width = info.Width / 4;
	int second_cropped_height = info.Height / 4;
	dim3 croppedGrids(iDivUp(second_cropped_width, TILE_DIM), iDivUp(second_cropped_height, TILE_DIM));

#if DEBUG
	float* d_SecondCropDebug;
	gpuErrorCheck(hipMalloc((float**)&d_SecondCropDebug, second_cropped_width * second_cropped_height * sizeof(float)));
	copyInterferenceComponentDebug << <croppedGrids, blocks >> > (d_magnitude, d_SecondCropDebug, max_loc_x, max_loc_y, info.HalfWidth, info.Height, second_cropped_width, second_cropped_height);
	gpuErrorCheck(hipDeviceSynchronize());

	float* h_SecondCropDebug = (float*)malloc(second_cropped_width * second_cropped_height * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_SecondCropDebug, d_SecondCropDebug, second_cropped_height * second_cropped_width * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_SecondCropDebug, second_cropped_width, second_cropped_height, false);
#endif
	
	fComplex* d_SecondCrop;
	gpuErrorCheck(hipMalloc((fComplex**)&d_SecondCrop, second_cropped_width * second_cropped_height * sizeof(fComplex)));
	copyInterferenceComponent << <croppedGrids, blocks >> > (d_Spectrum, d_SecondCrop, max_loc_x, max_loc_y, info.HalfWidth, info.Height, second_cropped_width, second_cropped_height);
	gpuErrorCheck(hipDeviceSynchronize());

	// ifft
	fComplex* d_rawWrapped;
	hipfftHandle ifftPlan;
	gpuErrorCheck(hipMalloc((fComplex**)&d_rawWrapped, second_cropped_width * second_cropped_height * sizeof(fComplex)));
	gpuErrorCheck(hipfftPlan2d(&ifftPlan, second_cropped_width, second_cropped_height, HIPFFT_C2C));
	gpuErrorCheck(hipfftExecC2C(ifftPlan, (hipfftComplex*)d_SecondCrop, (hipfftComplex*)d_rawWrapped, HIPFFT_BACKWARD));
	gpuErrorCheck(hipDeviceSynchronize());
	gpuErrorCheck(hipFree(d_SecondCrop));

	// test ifft
	//float* d_rawV;
	//gpuErrorCheck(hipMalloc((float**)&d_rawV, second_cropped_width * second_cropped_height * sizeof(float)));
	//complexToMagnitude << <croppedGrids, blocks >> > (d_rawWrapped, d_rawV, second_cropped_width, second_cropped_height);
	//gpuErrorCheck(hipDeviceSynchronize());
	//float* h_rawWrapped = (float*)malloc(second_cropped_width * second_cropped_height * sizeof(float));
	//gpuErrorCheck(hipMemcpy(h_rawWrapped, d_rawV, second_cropped_height * second_cropped_width * sizeof(float), hipMemcpyDeviceToHost));
	//displayImage(h_rawWrapped, second_cropped_width, second_cropped_height, false);

	//// arctan
	float* d_wrapped;
	gpuErrorCheck(hipMalloc((float**)&d_wrapped, second_cropped_width * second_cropped_height * sizeof(float)));
	applyArcTan << <croppedGrids, blocks >> > (d_rawWrapped, d_wrapped, second_cropped_width, second_cropped_height);
	gpuErrorCheck(hipDeviceSynchronize());

	float* h_wrapped = (float*)malloc(second_cropped_width * second_cropped_height * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_wrapped, d_wrapped, second_cropped_height * second_cropped_width * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_wrapped, second_cropped_width, second_cropped_height, false);
}

void getUnwrappedImage() {

}



//////
//#define IMAGE_DIM 256
//
//grid = dim3(IMAGE_DIM / 16, IMAGE_DIM / 16, 1);
//
//threads = dim3(16, 16, 1);
//
//// Declare handles to the FFT plans
//
//hipfftHandle forwardFFTPlan;
//
//hipfftHandle inverseFFTPlan;
//
//// Create the plans -- forward and reverse (Real2Complex, Complex2Real)
//
//CUFFT_SAFE_CALL(hipfftPlan2d(&forwardFFTPlan, IMAGE_DIM, IMAGE_DIM, HIPFFT_R2C));
//
//CUFFT_SAFE_CALL(hipfftPlan2d(&inverseFFTPlan, IMAGE_DIM, IMAGE_DIM, HIPFFT_C2R));
//
//int num_real_elements = IMAGE_DIM * IMAGE_DIM;
//
//int num_complex_elements = IMAGE_DIM * (IMAGE_DIM / 2 + 1);
//
//// HOST MEMORY
//
//float* h_img;
//
//float* h_imgF;
//
//// ALLOCATE HOST MEMORY
//
//h_img = (float*)malloc(m_num_real_elements * sizeof(float));
//
//h_complex_imgSpec = (hipfftComplex*)malloc(m_num_complex_elements * sizeof(hipfftComplex));
//
//h_imgF = (float*)malloc(m_num_real_elements * sizeof(float));
//
//for (int x = 0; x < IMAGE_DIM; x++)
//
//{
//
//	for (int y = 0; y < IMAGE_DIM; y++)
//
//	{
//
//		// initialize the input image memory somehow
//
//		// (this probably comes from a file or image buffer or something)
//
//		h_img[y * IMAGE_DIM + x] = 0.0f;
//
//	}
//
//}
//
//// DEVICE MEMORY
//
//float* d_img;
//
//hipfftComplex* d_complex_imgSpec;
//
//float* d_imgF;
//
//// ALLOCATE DEVICE MEMORY
//
//(hipMalloc((void**)&img, m_num_real_elements * sizeof(float)));
//
//(hipMalloc((void**)&d_complex_imgSpec, m_num_complex_elements * sizeof(hipfftComplex)));
//
//(hipMalloc((void**)&img, m_num_real_elements * sizeof(float)));
//
//// copy host memory to device (input image)
//
//(hipMemcpy(d_img, h_img, m_num_real_elements * sizeof(float), hipMemcpyHostToDevice));
//
//
//
//// now run the forward FFT on the device (real to complex)
//
//CUFFT_SAFE_CALL(hipfftExecR2C(forwardFFTPlan, d_img, d_complex_imgSpec));
//
//// copy the DEVICE complex data to the HOST
//
//// NOTE: we are only doing this so that you can see the data -- in general you want
//
//// to do your computation on the GPU without wasting the time of copying it back to the host
//
//(hipMemcpy(h_complex_imgSpec, d_complex_imgSpec, m_num_complex_elements * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
//
//// print the complex data so you can see what it looks like
//
//for (int x = 0; x < (IMAGE_DIM / 2 + 1); x++)
//
//{
//
//	for (int y = 0; y < IMAGE_DIM; y++)
//
//	{
//
//		// initialize the input image memory somehow
//
//		// (this probably comes from a file or image buffer or something)
//
//		printf("h_complex_imgSpec[%d,%d] = %f + %fi\n", x, y, h_complex_imgSpec[y * (IMAGE_DIM / 2 + 1) + x].x, h_complex_imgSpec[y * (IMAGE_DIM / 2 + 1) + x].y);
//
//	}
//
//}
//
//// here you can modify or filter the data in the frequency domain
//
//// TODO: insert your filter code here, or whatever
//
//// NOTE: you can/should modify it on the GPU/DEVICE, not the HOST
//
//// IF you modify it on the HOST you will need to hipMemcpy it back to the DEVICE
//
//// now run the inverse FFT on the device (complex to real)
//
//hipfftExecC2R(inverseFFTPlan, d_complex_imgSpec, d_imgF);
//
//// NOTE: the data in d_imgF is not normalized at this point
//
//// Normalize the data in place - IFFT Normalization is 
//
//// dividing all elements by the total numbers of elements in the matrix/image/array (ie, number of pixels)
//
//NormalizeIFFT << < grid, threads >> > (d_imgF, IMAGE_DIM, IMAGE_DIM, 256.0f * 256.0f);
//
//// Copy the DEVICE memory to the HOST memory
//
//(hipMemcpy(h_imgF, d_imgF, m_num_real_elements * sizeof(float), hipMemcpyDeviceToHost));
//
//// print the elements of the resulting data
//
//for (int i = 0; i < m_num_real_elements; i++)
//
//{
//
//	printf("h_imgF[%d] = %f\n", i, h_imgF[i]);
//
//}
//
//// CLEANUP HOST MEMORY
//
//free(h_img);
//
//free(h_imgF);
//
//// CLEANUP DEVICE MEMORY
//
//(hipFree(d_img));
//
//(hipFree(d_complex_imgSpec));
//
//(hipFree(d_imgF));