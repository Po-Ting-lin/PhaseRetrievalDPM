#include "hip/hip_runtime.h"
#include "phase_retriever.cuh"
#include <hipfft/hipfft.h>
#include <iostream>

#define D_NUM_STREAMS 8
#define DEBUG false
#define TIMER true

void phaseRetriever(cv::Mat& sp, cv::Mat& bg, float*& dst) {
	PhaseRetrieverInfo info;
	info.Image = nullptr;
	info.WrappedImage = nullptr;
	info.UnwrappedImage = nullptr;
	info.Width = sp.cols;
	info.Height = sp.rows;
	info.CroppedWidth = sp.cols / 4;
	info.CroppedHeight = sp.rows / 4;
	info.NumberOfRealElements = sp.cols * sp.rows;
	info.NumberOfCropElements = (sp.cols / 4) * (sp.rows / 4);
	info.Blocks = new dim3(TILE_DIM, TILE_DIM);
	info.Grids = new dim3(iDivUp(sp.cols, TILE_DIM), iDivUp(sp.rows, TILE_DIM));
	info.CroppedGrids = new dim3(iDivUp(sp.cols / 4, TILE_DIM), iDivUp(sp.rows / 4, TILE_DIM));
	
	float* sp_unwarpped = nullptr;
	float* bg_unwarpped = nullptr;
	processPhaseRetriever(sp, sp_unwarpped, info);
	processPhaseRetriever(bg, bg_unwarpped, info);

	for (int i = 0; i < info.NumberOfCropElements; i++) {
		sp_unwarpped[i] -= bg_unwarpped[i];
	}
	dst = sp_unwarpped;
	
	free(bg_unwarpped);
	delete info.Blocks;
	delete info.Grids;
	delete info.CroppedGrids;
}

void processPhaseRetriever(cv::Mat& src, float*& dst, PhaseRetrieverInfo& info) {
	info.Image = &src;

#if false
	auto t0 = std::chrono::system_clock::now();
#endif
	
	getWrappedImage(info);

#if false
	auto t1 = std::chrono::system_clock::now();
#endif
	getUnwrappedImage(info);

#if false
	auto t2 = std::chrono::system_clock::now();
	printTime(t0, t1, "getWrappedImage");
	printTime(t1, t2, "getUnwrappedImage");
#endif
	dst = info.UnwrappedImage;
}

void getWrappedImage(PhaseRetrieverInfo& info) {
	float* d_magnitude;
	float* d_firstCrop;
	uchar* image_ptr = info.Image->data;
	uchar* d_image_ptr;
	fComplex* d_data;
	fComplex* d_Spectrum;
	fComplex* d_SecondCrop;
	fComplex* d_rawWrapped;

	// make a FFT plan
	hipfftHandle fftPlan;
	hipfftHandle ifftPlan;
	gpuErrorCheck(hipfftPlan2d(&fftPlan, info.Height, info.Width, HIPFFT_C2C));
	gpuErrorCheck(hipfftPlan2d(&ifftPlan, info.CroppedWidth, info.CroppedHeight, HIPFFT_C2C));

	// aysn H to D
	hipStream_t stream[D_NUM_STREAMS];
	for (int i = 0; i < D_NUM_STREAMS; i++) {
		hipStreamCreate(&stream[i]);
	}
	gpuErrorCheck(hipMalloc((uchar**)&d_image_ptr, info.NumberOfRealElements * sizeof(uchar)));
	gpuErrorCheck(hipMalloc((fComplex**)&d_data, info.NumberOfRealElements * sizeof(fComplex)));
	int offset = 0;
	int data_elements_per_stream = info.NumberOfRealElements / D_NUM_STREAMS;
	int data_bytes_per_stream = info.NumberOfRealElements * sizeof(uchar) / D_NUM_STREAMS;
	dim3 grid(iDivUp(data_elements_per_stream, TILE_DIM));
	dim3 block(TILE_DIM);
	for (int i = 0; i < D_NUM_STREAMS; i++) {
		offset = i * data_elements_per_stream;
		gpuErrorCheck(hipMemcpyAsync(&d_image_ptr[offset], &image_ptr[offset], data_bytes_per_stream, hipMemcpyHostToDevice, stream[i]));
		realToComplex << <grid, block, 0, stream[i] >> > (&d_image_ptr[offset], &d_data[offset], info.NumberOfRealElements);
	}
	gpuErrorCheck(hipDeviceSynchronize());
	for (int i = 0; i < D_NUM_STREAMS; i++) {
		gpuErrorCheck(hipStreamDestroy(stream[i]));
	}

	// fft
	gpuErrorCheck(hipMalloc((fComplex**)&d_Spectrum, info.NumberOfRealElements * sizeof(fComplex)));
	gpuErrorCheck(hipfftExecC2C(fftPlan, (hipfftComplex*)d_data, (hipfftComplex*)d_Spectrum, HIPFFT_FORWARD));
	gpuErrorCheck(hipDeviceSynchronize());

	// magnitude
	gpuErrorCheck(hipMalloc((float**)&d_magnitude, info.NumberOfRealElements * sizeof(float)));
	complexToMagnitude << <*info.Grids, *info.Blocks >> > (d_Spectrum, d_magnitude, info.Width, info.Height);
	gpuErrorCheck(hipDeviceSynchronize());

#if DEBUG
	float* h_magnitude = (float*)malloc(info.NumberOfRealElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_magnitude, d_magnitude, info.NumberOfRealElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_magnitude, info.Width, info.Height, true);
#endif

	// find max index
	gpuErrorCheck(hipMalloc((float**)&d_firstCrop, info.NumberOfCropElements * sizeof(float)));
	copyInterferenceComponentRoughly << <*info.CroppedGrids, *info.Blocks >> > (d_magnitude, d_firstCrop, info.Width, info.Height, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

#if DEBUG
	float* h_firstCrop = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_firstCrop, d_firstCrop, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_firstCrop, info.CroppedWidth, info.CroppedHeight, true);
#endif

	thrust::device_ptr<float> d_ptr(d_firstCrop);
	thrust::device_vector<float> d_vec(d_ptr, d_ptr + info.NumberOfCropElements);
	thrust::device_vector<float>::iterator iter = thrust::max_element(d_vec.begin(), d_vec.end());
	unsigned int index = iter - d_vec.begin();
	float max_val = *iter;
	int max_cropped_x = index % info.CroppedWidth;
	int max_cropped_y = index / info.CroppedWidth;
	int max_x = max_cropped_x < info.CroppedWidth / 2 ? max_cropped_x + 7 * info.Width / 8 : max_cropped_x - info.CroppedWidth / 2;
	int max_y = max_cropped_y + info.Height / 2;
	//std::cout << "Position is x: " << max_loc_x << "y: " << max_loc_y << std::endl;

	// crop
#if DEBUG
	float* d_SecondCropDebug;
	gpuErrorCheck(hipMalloc((float**)&d_SecondCropDebug, info.NumberOfCropElements * sizeof(float)));
	copyInterferenceComponentDebug << <*info.CroppedGrids, *info.Blocks >> > (d_magnitude, d_SecondCropDebug, max_x, max_y, info.Width, info.Height, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	float* h_SecondCropDebug = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_SecondCropDebug, d_SecondCropDebug, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_SecondCropDebug, info.CroppedWidth, info.CroppedHeight, false);
#endif
	gpuErrorCheck(hipMalloc((fComplex**)&d_SecondCrop, info.NumberOfCropElements * sizeof(fComplex)));
	copyInterferenceComponent << <*info.CroppedGrids, *info.Blocks >> > (d_Spectrum, d_SecondCrop, max_x, max_y, info.Width, info.Height, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	// ifft
	gpuErrorCheck(hipMalloc((fComplex**)&d_rawWrapped, info.NumberOfCropElements * sizeof(fComplex)));
	gpuErrorCheck(hipfftExecC2C(ifftPlan, (hipfftComplex*)d_SecondCrop, (hipfftComplex*)d_rawWrapped, HIPFFT_BACKWARD));
	gpuErrorCheck(hipDeviceSynchronize());

	// arctan
	gpuErrorCheck(hipMalloc((float**)&info.WrappedImage, info.NumberOfCropElements * sizeof(float)));
	applyArcTan << <*info.CroppedGrids, *info.Blocks >> > (d_rawWrapped, info.WrappedImage, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	// free
	gpuErrorCheck(hipfftDestroy(fftPlan));
	gpuErrorCheck(hipfftDestroy(ifftPlan));
	gpuErrorCheck(hipFree(d_data));
	gpuErrorCheck(hipFree(d_magnitude));
	gpuErrorCheck(hipFree(d_firstCrop));
	gpuErrorCheck(hipFree(d_SecondCrop));
	gpuErrorCheck(hipFree(d_rawWrapped));
}

void getUnwrappedImage(PhaseRetrieverInfo& info) {
#if DEBUG
	float* h_wrapped_image = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_wrapped_image, info.WrappedImage, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	displayImage(h_wrapped_image, info.CroppedWidth, info.CroppedHeight, false);
	free(h_wrapped_image);
#endif

	float* d_dx;
	float* d_dy;
	float* d_sumC;
	float* d_divider;
	gpuErrorCheck(hipMalloc((float**)&d_dx, info.NumberOfCropElements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_dy, info.NumberOfCropElements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_sumC, info.NumberOfCropElements * sizeof(float)));
	gpuErrorCheck(hipMalloc((float**)&d_divider, info.NumberOfCropElements * sizeof(float)));
	float tx = PI / info.CroppedWidth;
	float ty = PI / info.CroppedHeight;
	// diff
	applyDifference<<<*info.CroppedGrids, *info.Blocks >>>(info.WrappedImage, d_dx, d_dy, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	
#if DEBUG
	float* h_dx = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_dx, d_dx, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	float* h_dy = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_dy, d_dy, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	//displayImage(h_dx, info.CroppedWidth, info.CroppedHeight, false);

	std::cout << "***********************dx section***********************" << std::endl;
	for (int i = 0; i < 10; i++) {
		std::cout << "i: " << i << "; value: " << h_dx[i] << std::endl;
	}
	for (int i = info.CroppedWidth - 10; i < info.CroppedWidth; i++) {
		std::cout << "i: " << i << "; value: " << h_dx[i] << std::endl;
	}
	std::cout << "***********************dy section***********************" << std::endl;
	for (int i = 0; i < 10; i++) {
		std::cout << "i: " << i << "; value: " << h_dy[i] << std::endl;
	}
	for (int i = info.CroppedWidth - 10; i < info.CroppedWidth; i++) {
		std::cout << "i: " << i << "; value: " << h_dy[i] << std::endl;
	}
	free(h_dx);
	free(h_dy);
#endif
	
	applySum << <*info.CroppedGrids, *info.Blocks >> > (d_dx, d_dy, d_sumC, d_divider, tx, ty, info.CroppedWidth, info.CroppedHeight);
	gpuErrorCheck(hipDeviceSynchronize());

	gpuErrorCheck(hipFree(d_dx));
	gpuErrorCheck(hipFree(d_dy));

	float* h_sumC = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_sumC, d_sumC, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));
	float* h_divider = (float*)malloc(info.NumberOfCropElements * sizeof(float));
	gpuErrorCheck(hipMemcpy(h_divider, d_divider, info.NumberOfCropElements * sizeof(float), hipMemcpyDeviceToHost));

#if DEBUG
	std::cout << "***********************sumC section***********************" << std::endl;
	for (int i = 0; i < 10; i++) {
		std::cout << "i: " << i << "; value: " << h_sumC[i] << std::endl;
	}
	for (int i = info.CroppedWidth - 10; i < info.CroppedWidth; i++) {
		std::cout << "i: " << i << "; value: " << h_sumC[i] << std::endl;
	}
	std::cout << "***********************divider section***********************" << std::endl;
	for (int i = 0; i < 10; i++) {
		std::cout << "i: " << i << "; value: " << h_divider[i] << std::endl;
	}
	for (int i = info.CroppedWidth - 10; i < info.CroppedWidth; i++) {
		std::cout << "i: " << i << "; value: " << h_divider[i] << std::endl;
	}
#endif
	cv::Mat sumC_mat(info.CroppedHeight, info.CroppedWidth, CV_32FC1, h_sumC);
	cv::dct(sumC_mat, sumC_mat, cv::DCT_ROWS);
	cv::transpose(sumC_mat, sumC_mat);
	cv::dct(sumC_mat, sumC_mat, cv::DCT_ROWS);
	cv::transpose(sumC_mat, sumC_mat);
	
#if DEBUG
	for (int i = 0; i < 10; i++) {
		std::cout << "i: " << i << "; value: " << h_sumC[i] << std::endl;
	}
	for (int i = 0 + info.CroppedWidth; i < 10 + info.CroppedWidth; i++) {
		std::cout << "i: " << i << "; value: " << h_sumC[i] << std::endl;
	}
#endif
	
	// divide, can be optimized
	for (int i = 0; i < info.NumberOfCropElements; i++) {
		if (h_divider[i] != 0.0f) {
			h_sumC[i] = h_sumC[i] / h_divider[i];
		}
	}

	// iDCT
	cv::idct(sumC_mat, sumC_mat, cv::DCT_ROWS);
	cv::transpose(sumC_mat, sumC_mat);
	cv::idct(sumC_mat, sumC_mat, cv::DCT_ROWS);
	cv::transpose(sumC_mat, sumC_mat);

#if DEBUG
	for (int i = 0; i < 10; i++) {
		std::cout << "i: " << i << "; value: " << h_sumC[i] << std::endl;
	}
	for (int i = 0 + info.CroppedWidth; i < 10 + info.CroppedWidth; i++) {
		std::cout << "i: " << i << "; value: " << h_sumC[i] << std::endl;
	}
	displayImage(h_sumC, info.CroppedWidth, info.CroppedHeight, false);
#endif

	info.UnwrappedImage = h_sumC;

	// free
	free(h_divider);
	gpuErrorCheck(hipFree(d_sumC));
	gpuErrorCheck(hipFree(d_divider));
	gpuErrorCheck(hipFree(info.WrappedImage));
}
